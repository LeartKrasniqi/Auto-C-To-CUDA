
#include <hip/hip_runtime.h>
#define CUDA_BLOCK_X 128
#define CUDA_BLOCK_Y 1
#define CUDA_BLOCK_Z 1

__global__ void _auto_kernel_0(int a[100][2])
{
  int thread_x_id;thread_x_id = blockIdx.x * blockDim.x + threadIdx.x;
  int thread_y_id;thread_y_id = blockIdx.y * blockDim.y + threadIdx.y;
  if (thread_x_id && thread_y_id) 
    if (thread_x_id <= 100 && thread_y_id <= 2) {
      a[2 * thread_x_id + -2][2 * thread_y_id + -2] = a[2 * thread_x_id + -1][2 * thread_y_id + -1];
    }
}

int main()
{
  int a[100][2];
  int i;
  int j;
  i = 0;
{
{
/* Auto-generated code for call to _auto_kernel_0 */
      typedef int _narray_a[2];
      _narray_a *d_a;
    hipMalloc((void **) &d_a, sizeof(int ) * 100 * 2);
    hipMemcpy(d_a, a, sizeof(int ) * 100 * 2, hipMemcpyHostToDevice);
      int CUDA_GRID_X;
    CUDA_GRID_X = (100 + CUDA_BLOCK_X - 1)/CUDA_BLOCK_X;
      int CUDA_GRID_Y;
    CUDA_GRID_Y = (2 + CUDA_BLOCK_Y - 1)/CUDA_BLOCK_Y;
      int CUDA_GRID_Z;
    CUDA_GRID_Z = (1 + CUDA_BLOCK_Z - 1)/CUDA_BLOCK_Z;
    const dim3 CUDA_blockSize(CUDA_BLOCK_X, CUDA_BLOCK_Y, CUDA_BLOCK_Z);
    const dim3 CUDA_gridSize(CUDA_GRID_X, CUDA_GRID_Y, CUDA_GRID_Z);
    _auto_kernel_0<<<CUDA_gridSize,CUDA_blockSize>>>(d_a);
    hipMemcpy(a, d_a, sizeof(int ) * 100 * 2, hipMemcpyDeviceToHost);
    }
  }
  return 2;
}
