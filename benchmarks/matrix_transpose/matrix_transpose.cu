#include "hip/hip_runtime.h"
/* Simple program to obtain transpose of a matrix */
#include <stdio.h>
#define CUDA_BLOCK_X 128
#define CUDA_BLOCK_Y 1
#define CUDA_BLOCK_Z 1

__global__ void _auto_kernel_0(int m,int n,char A[m][n],char B[n][m])
{
  int thread_x_id;thread_x_id = blockIdx.x * blockDim.x + threadIdx.x;
  int thread_y_id;thread_y_id = blockIdx.y * blockDim.y + threadIdx.y;
  if (thread_x_id && thread_y_id) 
    if (thread_x_id <= (m + 0) / 1 && thread_y_id <= (n + 0) / 1) {
      B[1 * thread_y_id + -1][1 * thread_x_id + -1] = A[1 * thread_x_id + -1][1 * thread_y_id + -1];
    }
}

int main()
{
  int j_nom_4;
  int i_nom_3;
  int j_nom_2;
  int i_nom_1;
  int j;
  int i;
  int m = 1000;
  int n = 500;
  char A[m][n];
  char B[n][m];
/* Initialize */
  srand((time(0)));
  for (i = 1; i <= (m + 0) / 1; i += 1) {
    for (j = 1; j <= (n + 0) / 1; j += 1) {
      A[1 * i + -1][1 * j + -1] = (rand() % 25 + 'A');
    }
  }
{
/* Auto-generated code for call to _auto_kernel_0 */
    typedef char _narray_A[n];
    _narray_A *d_A;
    hipMalloc((void **) &d_A, sizeof(char ) * m * n);
    hipMemcpy(d_A, A, sizeof(char ) * m * n, hipMemcpyHostToDevice);
    typedef char _narray_B[m];
    _narray_B *d_B;
    hipMalloc((void **) &d_B, sizeof(char ) * n * m);
    hipMemcpy(d_B, B, sizeof(char ) * n * m, hipMemcpyHostToDevice);
    int CUDA_GRID_X;
    CUDA_GRID_X = (1 + CUDA_BLOCK_X - 1)/CUDA_BLOCK_X;
    int CUDA_GRID_Y;
    CUDA_GRID_Y = (1 + CUDA_BLOCK_Y - 1)/CUDA_BLOCK_Y;
    int CUDA_GRID_Z;
    CUDA_GRID_Z = (1 + CUDA_BLOCK_Z - 1)/CUDA_BLOCK_Z;
    const dim3 CUDA_blockSize(CUDA_BLOCK_X, CUDA_BLOCK_Y, CUDA_BLOCK_Z);
    const dim3 CUDA_gridSize(CUDA_GRID_X, CUDA_GRID_Y, CUDA_GRID_Z);
    _auto_kernel_0<<<CUDA_gridSize,CUDA_blockSize>>>(m, n, d_A, d_B);
    hipMemcpy(A, d_A, sizeof(char ) * m * n, hipMemcpyDeviceToHost);
    hipMemcpy(B, d_B, sizeof(char ) * n * m, hipMemcpyDeviceToHost);
  }
/* Assertion */
  for (i_nom_3 = 1; i_nom_3 <= (m + 0) / 1; i_nom_3 += 1) {
    for (j_nom_4 = 1; j_nom_4 <= (n + 0) / 1; j_nom_4 += 1) {
      if (A[1 * i_nom_3 + -1][1 * j_nom_4 + -1] != B[1 * j_nom_4 + -1][1 * i_nom_3 + -1]) {
        fprintf(stderr,"ERROR\n");
        exit(-1);
      }
    }
  }
  printf("All good b0ss\n");
  return 0;
}
