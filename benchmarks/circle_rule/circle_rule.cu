/* Computes quadrature rules (i.e. circumference) for unit circle in 2D */
/* Adapted from: https://people.sc.fsu.edu/~jburkardt/c_src/circle_rule/circle_rule.html */

#include <hip/hip_runtime.h>
#include <stdio.h>
#define NUM_ANGLES 100000
#define PI 3.14159265358
#define F(x,y) x*y
#define CUDA_BLOCK_X 128
#define CUDA_BLOCK_Y 1
#define CUDA_BLOCK_Z 1

__global__ void _auto_kernel_1(float w[100000],float Q[100000],float x[100000],float y[100000])
{
  int thread_x_id;thread_x_id = blockIdx.x * blockDim.x + threadIdx.x;
  if (thread_x_id) 
    if (thread_x_id <= 100000) {
      Q[1 * thread_x_id + -1] = w[1 * thread_x_id + -1] * x[1 * thread_x_id + -1] * y[1 * thread_x_id + -1];
    }
}

__global__ void _auto_kernel_0(float w[100000],float a[100000])
{
  int thread_x_id;thread_x_id = blockIdx.x * blockDim.x + threadIdx.x;
  if (thread_x_id) 
    if (thread_x_id <= 100000) {
      w[1 * thread_x_id + -1] = ((float )(1.0 / ((double )((float )100000))));
      a[1 * thread_x_id + -1] = ((float )(6.28319 * ((double )((float )(1 * thread_x_id + -1))) / ((double )((float )100000))));
    }
}

int main()
{
  int i_nom_3;
  int i_nom_2;
  int i_nom_1;
  int i;
/* Weights */
  float w[100000];
/* Angles */
  float a[100000];
/* Result */
  float Q[100000];
{
/* Auto-generated code for call to _auto_kernel_0 */
    typedef float _narray_w;
    _narray_w *d_w;
    hipMalloc((void **) &d_w, sizeof(float ) * 100000);
    hipMemcpy(d_w, w, sizeof(float ) * 100000, hipMemcpyHostToDevice);
    typedef float _narray_a;
    _narray_a *d_a;
    hipMalloc((void **) &d_a, sizeof(float ) * 100000);
    hipMemcpy(d_a, a, sizeof(float ) * 100000, hipMemcpyHostToDevice);
    int CUDA_GRID_X;
    CUDA_GRID_X = (100000 + CUDA_BLOCK_X - 1)/CUDA_BLOCK_X;
    int CUDA_GRID_Y;
    CUDA_GRID_Y = (1 + CUDA_BLOCK_Y - 1)/CUDA_BLOCK_Y;
    int CUDA_GRID_Z;
    CUDA_GRID_Z = (1 + CUDA_BLOCK_Z - 1)/CUDA_BLOCK_Z;
    const dim3 CUDA_blockSize(CUDA_BLOCK_X, CUDA_BLOCK_Y, CUDA_BLOCK_Z);
    const dim3 CUDA_gridSize(CUDA_GRID_X, CUDA_GRID_Y, CUDA_GRID_Z);
    _auto_kernel_0<<<CUDA_gridSize,CUDA_blockSize>>>(d_w, d_a);
    hipMemcpy(w, d_w, sizeof(float ) * 100000, hipMemcpyDeviceToHost);
    hipMemcpy(a, d_a, sizeof(float ) * 100000, hipMemcpyDeviceToHost);
  }
/* Useful sin/cos values */
  float x[100000];
  float y[100000];
  for (i_nom_1 = 1; i_nom_1 <= 100000; i_nom_1 += 1) {
    x[1 * i_nom_1 + -1] = (cos(a[1 * i_nom_1 + -1]));
    y[1 * i_nom_1 + -1] = (sin(a[1 * i_nom_1 + -1]));
  }
{
/* Auto-generated code for call to _auto_kernel_1 */
    typedef float _narray_w;
    _narray_w *d_w;
    hipMalloc((void **) &d_w, sizeof(float ) * 100000);
    hipMemcpy(d_w, w, sizeof(float ) * 100000, hipMemcpyHostToDevice);
    typedef float _narray_Q;
    _narray_Q *d_Q;
    hipMalloc((void **) &d_Q, sizeof(float ) * 100000);
    hipMemcpy(d_Q, Q, sizeof(float ) * 100000, hipMemcpyHostToDevice);
    typedef float _narray_x;
    _narray_x *d_x;
    hipMalloc((void **) &d_x, sizeof(float ) * 100000);
    hipMemcpy(d_x, x, sizeof(float ) * 100000, hipMemcpyHostToDevice);
    typedef float _narray_y;
    _narray_y *d_y;
    hipMalloc((void **) &d_y, sizeof(float ) * 100000);
    hipMemcpy(d_y, y, sizeof(float ) * 100000, hipMemcpyHostToDevice);
    int CUDA_GRID_X;
    CUDA_GRID_X = (100000 + CUDA_BLOCK_X - 1)/CUDA_BLOCK_X;
    int CUDA_GRID_Y;
    CUDA_GRID_Y = (1 + CUDA_BLOCK_Y - 1)/CUDA_BLOCK_Y;
    int CUDA_GRID_Z;
    CUDA_GRID_Z = (1 + CUDA_BLOCK_Z - 1)/CUDA_BLOCK_Z;
    const dim3 CUDA_blockSize(CUDA_BLOCK_X, CUDA_BLOCK_Y, CUDA_BLOCK_Z);
    const dim3 CUDA_gridSize(CUDA_GRID_X, CUDA_GRID_Y, CUDA_GRID_Z);
    _auto_kernel_1<<<CUDA_gridSize,CUDA_blockSize>>>(d_w, d_Q, d_x, d_y);
    hipMemcpy(w, d_w, sizeof(float ) * 100000, hipMemcpyDeviceToHost);
    hipMemcpy(Q, d_Q, sizeof(float ) * 100000, hipMemcpyDeviceToHost);
    hipMemcpy(x, d_x, sizeof(float ) * 100000, hipMemcpyDeviceToHost);
    hipMemcpy(y, d_y, sizeof(float ) * 100000, hipMemcpyDeviceToHost);
  }
  double sum = 0;
  for (i_nom_3 = 1; i_nom_3 <= 100000; i_nom_3 += 1) {
    sum += Q[1 * i_nom_3 + -1];
  }
  double result = 2 * 3.14159265358 * sum;
/* Report the result */
  printf("Result: %f\n",result);
  return 0;
}
