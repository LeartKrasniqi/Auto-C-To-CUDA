#include "hip/hip_runtime.h"
/* Calculate the Discrete Fourier Transform of a signal */
/* Adapted from: https://batchloaf.wordpress.com/2013/12/07/simple-dft-in-c/ */
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
/* Assume N is greater than 4 and a power of 2 */
#define N 64
#define PI2 6.2832
#define CUDA_BLOCK_X 128
#define CUDA_BLOCK_Y 1
#define CUDA_BLOCK_Z 1

__global__ void _auto_kernel_2(float x_re[33],float x_im[33],float P[33])
{
  int thread_x_id;thread_x_id = blockIdx.x * blockDim.x + threadIdx.x;
  if (thread_x_id) 
    if (thread_x_id <= 33) {
      P[1 * thread_x_id + -1] = x_re[1 * thread_x_id + -1] * x_re[1 * thread_x_id + -1] + x_im[1 * thread_x_id + -1] * x_im[1 * thread_x_id + -1];
    }
}

__global__ void _auto_kernel_1(float sin_vals[64][64],float x[64],float x_im[33])
{
  int thread_x_id;thread_x_id = blockIdx.x * blockDim.x + threadIdx.x;
  int thread_y_id;thread_y_id = blockIdx.y * blockDim.y + threadIdx.y;
  if (thread_x_id && thread_y_id) 
    if (thread_x_id <= 33 && thread_y_id <= 64) {
//x_im_inter[n];
      x_im[1 * thread_x_id + -1] -= x[1 * thread_y_id + -1] * sin_vals[1 * thread_x_id + -1][1 * thread_x_id + -1];
    }
}

__global__ void _auto_kernel_0(float cos_vals[64][64],float x[64],float x_re[33])
{
  int thread_x_id;thread_x_id = blockIdx.x * blockDim.x + threadIdx.x;
  int thread_y_id;thread_y_id = blockIdx.y * blockDim.y + threadIdx.y;
  if (thread_x_id && thread_y_id) 
    if (thread_x_id <= 33 && thread_y_id <= 64) {
//x_re_inter[n];
      x_re[1 * thread_x_id + -1] += x[1 * thread_y_id + -1] * cos_vals[1 * thread_x_id + -1][1 * thread_y_id + -1];
    }
}

int main()
{
  int i;
  int j;
  int n;
  int k;
/* Values of sin and cos */
  float sin_vals[64][64];
  float cos_vals[64][64];
  for (i = 1; i <= 64; i += 1) {
    for (j = 1; j <= 64; j += 1) {
      sin_vals[1 * i + -1][1 * j + -1] = (sin(((1 * i + -1) * (1 * j + -1)) * 6.2832 / 64));
      cos_vals[1 * i + -1][1 * j + -1] = (cos(((1 * i + -1) * (1 * j + -1)) * 6.2832 / 64));
    }
  }
/* Discrete time signal -- Generate a random signal in range (-1, 1) */
  float x[64];
  srand((time(0)));
  for (i = 1; i <= 64; i += 1) {
    x[1 * i + -1] = (2.0 * (rand()) / 2147483647 - 1.0 + sin(6.2832 * (1 * i + -1) * 5.7 / 64));
  }
/* These will hold the DFT x (both real and imaginary parts) */
  float x_re[33];
  float x_im[33];
/* This will hold the power spectrum of x */
  float P[33];
{
{
{
/* Auto-generated code for call to _auto_kernel_0 */
        typedef float _narray_cos_vals[64];
        _narray_cos_vals *d_cos_vals;
    hipMalloc((void **) &d_cos_vals, sizeof(float ) * 64 * 64);
    hipMemcpy(d_cos_vals, cos_vals, sizeof(float ) * 64 * 64, hipMemcpyHostToDevice);
        typedef float _narray_x;
        _narray_x *d_x;
    hipMalloc((void **) &d_x, sizeof(float ) * 64);
    hipMemcpy(d_x, x, sizeof(float ) * 64, hipMemcpyHostToDevice);
        typedef float _narray_x_re;
        _narray_x_re *d_x_re;
    hipMalloc((void **) &d_x_re, sizeof(float ) *(32 + 1));
    hipMemcpy(d_x_re, x_re, sizeof(float ) *(32 + 1), hipMemcpyHostToDevice);
        int CUDA_GRID_X;
    CUDA_GRID_X = (64 + CUDA_BLOCK_X - 1)/CUDA_BLOCK_X;
        int CUDA_GRID_Y;
    CUDA_GRID_Y = (64 + CUDA_BLOCK_Y - 1)/CUDA_BLOCK_Y;
        int CUDA_GRID_Z;
    CUDA_GRID_Z = (1 + CUDA_BLOCK_Z - 1)/CUDA_BLOCK_Z;
    const dim3 CUDA_blockSize(CUDA_BLOCK_X, CUDA_BLOCK_Y, CUDA_BLOCK_Z);
    const dim3 CUDA_gridSize(CUDA_GRID_X, CUDA_GRID_Y, CUDA_GRID_Z);
    _auto_kernel_0<<<CUDA_gridSize,CUDA_blockSize>>>(d_cos_vals, d_x, d_x_re);
    hipMemcpy(cos_vals, d_cos_vals, sizeof(float ) * 64 * 64, hipMemcpyDeviceToHost);
    hipMemcpy(x, d_x, sizeof(float ) * 64, hipMemcpyDeviceToHost);
    hipMemcpy(x_re, d_x_re, sizeof(float ) *(32 + 1), hipMemcpyDeviceToHost);
      }
    }
{
{
/* Auto-generated code for call to _auto_kernel_1 */
        typedef float _narray_sin_vals[64];
        _narray_sin_vals *d_sin_vals;
    hipMalloc((void **) &d_sin_vals, sizeof(float ) * 64 * 64);
    hipMemcpy(d_sin_vals, sin_vals, sizeof(float ) * 64 * 64, hipMemcpyHostToDevice);
        typedef float _narray_x;
        _narray_x *d_x;
    hipMalloc((void **) &d_x, sizeof(float ) * 64);
    hipMemcpy(d_x, x, sizeof(float ) * 64, hipMemcpyHostToDevice);
        typedef float _narray_x_im;
        _narray_x_im *d_x_im;
    hipMalloc((void **) &d_x_im, sizeof(float ) *(32 + 1));
    hipMemcpy(d_x_im, x_im, sizeof(float ) *(32 + 1), hipMemcpyHostToDevice);
        int CUDA_GRID_X;
    CUDA_GRID_X = (64 + CUDA_BLOCK_X - 1)/CUDA_BLOCK_X;
        int CUDA_GRID_Y;
    CUDA_GRID_Y = (64 + CUDA_BLOCK_Y - 1)/CUDA_BLOCK_Y;
        int CUDA_GRID_Z;
    CUDA_GRID_Z = (1 + CUDA_BLOCK_Z - 1)/CUDA_BLOCK_Z;
    const dim3 CUDA_blockSize(CUDA_BLOCK_X, CUDA_BLOCK_Y, CUDA_BLOCK_Z);
    const dim3 CUDA_gridSize(CUDA_GRID_X, CUDA_GRID_Y, CUDA_GRID_Z);
    _auto_kernel_1<<<CUDA_gridSize,CUDA_blockSize>>>(d_sin_vals, d_x, d_x_im);
    hipMemcpy(sin_vals, d_sin_vals, sizeof(float ) * 64 * 64, hipMemcpyDeviceToHost);
    hipMemcpy(x, d_x, sizeof(float ) * 64, hipMemcpyDeviceToHost);
    hipMemcpy(x_im, d_x_im, sizeof(float ) *(32 + 1), hipMemcpyDeviceToHost);
      }
    }
{
/* Auto-generated code for call to _auto_kernel_2 */
      typedef float _narray_x_re;
      _narray_x_re *d_x_re;
    hipMalloc((void **) &d_x_re, sizeof(float ) *(32 + 1));
    hipMemcpy(d_x_re, x_re, sizeof(float ) *(32 + 1), hipMemcpyHostToDevice);
      typedef float _narray_x_im;
      _narray_x_im *d_x_im;
    hipMalloc((void **) &d_x_im, sizeof(float ) *(32 + 1));
    hipMemcpy(d_x_im, x_im, sizeof(float ) *(32 + 1), hipMemcpyHostToDevice);
      typedef float _narray_P;
      _narray_P *d_P;
    hipMalloc((void **) &d_P, sizeof(float ) *(32 + 1));
    hipMemcpy(d_P, P, sizeof(float ) *(32 + 1), hipMemcpyHostToDevice);
      int CUDA_GRID_X;
    CUDA_GRID_X = (1 + CUDA_BLOCK_X - 1)/CUDA_BLOCK_X;
      int CUDA_GRID_Y;
    CUDA_GRID_Y = (1 + CUDA_BLOCK_Y - 1)/CUDA_BLOCK_Y;
      int CUDA_GRID_Z;
    CUDA_GRID_Z = (1 + CUDA_BLOCK_Z - 1)/CUDA_BLOCK_Z;
    const dim3 CUDA_blockSize(CUDA_BLOCK_X, CUDA_BLOCK_Y, CUDA_BLOCK_Z);
    const dim3 CUDA_gridSize(CUDA_GRID_X, CUDA_GRID_Y, CUDA_GRID_Z);
    _auto_kernel_2<<<CUDA_gridSize,CUDA_blockSize>>>(d_x_re, d_x_im, d_P);
    hipMemcpy(x_re, d_x_re, sizeof(float ) *(32 + 1), hipMemcpyDeviceToHost);
    hipMemcpy(x_im, d_x_im, sizeof(float ) *(32 + 1), hipMemcpyDeviceToHost);
    hipMemcpy(P, d_P, sizeof(float ) *(32 + 1), hipMemcpyDeviceToHost);
    }
  }
  return 0;
}
